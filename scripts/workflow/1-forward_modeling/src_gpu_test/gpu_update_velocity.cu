#include "hip/hip_runtime.h"
#include <stdio.h>
#include "common.h"


__global__ void  kernel_update_velocity(float* vx, float* vz, float* txx, float* tzz, float* txz,\
	float* C, float* pml_xtxx, float* pml_xtxz, float* pml_ztxz, float* pml_ztzz, float* pml_para,\
  int*  rec_loc, int* src_loc, float wave,\
	int NZ, int NX, int NK, int NR, int nabs, int fnabs,\
	float dx, float dz, float dt, int it){


  //printf("device vx[100] = %f\n", vx[100]);
  //vx[100] = 13;
  //printf("device vx[100] = %f\n", vx[100]);

	//local parameters
	int ix, iz, pos, posc, ipml, zcom, xcom, pmlpos;
	float xtxx, xtxz, ztxz, ztzz, Ttmp;
	float dtdx, dtdz;

	dtdx = dt/dx;
  dtdz = dt/dz;
	//add source
  // vz[ (src_loc[0] + nabs) * NZ + src_loc[1] + fnabs] += wave;
	// vx[ (src_loc[0] + nabs) * NZ + src_loc[1] + fnabs] += wave;
  
  ix = blockIdx.x * blockDim.x + threadIdx.x;
  iz = blockIdx.y * blockDim.y + threadIdx.y;
  //printf("%d\n", threadIdx.x);
  //printf("it %d\n", it);
  //printf("ix %d\n", ix);
  //printf("NX %d NZ %d in kernel_velocity\n", NX, NZ);
  if(ix > 0 &&  ix < NX - 1 && iz > 0 && iz < NZ - 1){
  //printf("ix %d\n", ix);
	// calculate derivative;
	//for(ix = 1; ix < NX - 1; ix++) 
	//    for(iz = 1; iz < NZ - 1; iz++) {
	    	pos = ix * NZ + iz;
	    	posc = ix * NZ * NK + iz * NK;

			if ( ix == 1 || ix == NX - 2 ) {
			xtxx = txx[pos] - txx[pos - NZ];
			xtxz = txz[pos + NZ] - txz[pos];
			}
			else {
			xtxx = coe1 * (txx[pos]  - txx[pos - NZ]) + coe2 * (txx[pos + NZ] - txx[pos - 2*NZ]);
			xtxz = coe1 * (txz[pos + NZ] - txz[pos])   + coe2 * (txz[pos + 2*NZ] - txz[pos - NZ]);
			}

			if( iz == 1 || iz == NZ - 2 )  {
			ztxz = txz[pos] - txz[pos - 1];
			ztzz = tzz[pos + 1] - tzz[pos];
			}
			else {
			ztxz = coe1 * (txz[pos]  - txz[pos - 1]) + coe2 * (txz[pos + 1] - txz[pos - 2]);
			ztzz=  coe1 * (tzz[pos + 1] - tzz[pos])  + coe2 * (tzz[pos + 2] - tzz[pos - 1]);
			}
			// dtdx = dt/dx
			vx[pos] += (xtxx * dtdx + ztxz * dtdz) * 2.0 / (C[posc + 0] + C[posc - NZ*NK + 0]);
			vz[pos] += (xtxz * dtdx + ztzz * dtdz) * 2.0 / (C[posc + 0] + C[posc + NK + 0]);
      
      //__syncthreads();

			zcom =  ((ix >= nabs) && (ix < NX - nabs) && (iz >= NZ - nabs)) ? 1:0;
			xcom =  (ix <= nabs) ? 0: ( (ix < NX - nabs) ? (ix - nabs) : (NX - 2*nabs));
			pmlpos = pos - ( NZ - fnabs - nabs) * (zcom + xcom);

		// PML at X direction
			if(ix < nabs) {
			 ipml = nabs - 1 - ix;
			 // vx
			 Ttmp = (2.0 * pml_xtxx[pmlpos] + pml_para[ipml*12+4] * xtxx) / (2.0 + pml_para[ipml*12+0]);
			 vx[pos] += ((pml_para[ipml*12+2] - 1) * xtxx / dx - pml_para[ipml*12+2] * Ttmp) * dt * 2.0/(C[posc + 0] + C[posc - NZ*NK + 0]);
			 pml_xtxx[pmlpos] = 2.0 * Ttmp - pml_xtxx[pmlpos];
			 // vz
			 Ttmp = (2.0 * pml_xtxz[pmlpos] + pml_para[ipml*12+5] * xtxz) / (2.0 + pml_para[ipml*12+1]);
			 //vz[pos] += ((pml_beta_half[ipml] - 1) * xtxz / dx - pml_beta_half[ipml] * Ttmp) * dt * 2.0/(C[posc + 0] + C[posc + NK + 0]);
			 vz[pos] += ((pml_para[ipml*12+3] - 1) * xtxz / dx - pml_para[ipml*12+3] * Ttmp) * dt * 2.0/(C[posc + 0] + C[posc + NK + 0]);
			 pml_xtxz[pmlpos] = 2.0 * Ttmp - pml_xtxz[pmlpos];
			}
			if(ix >= NX - nabs) {
			 // vz
			 ipml = ix - (NX - nabs);
			 Ttmp = (2.0 * pml_xtxz[pmlpos] + pml_para[ipml*12+5] * xtxz) / (2.0 + pml_para[ipml*12+1]);
			 //vz[pos] += ((pml_beta_half[ipml] - 1) * xtxz / dx - pml_beta_half[ipml] * Ttmp) * dt * 2.0/(C[posc + 0] + C[posc + NK + 0]);
			 vz[pos] += ((pml_para[ipml*12+3] - 1) * xtxz / dx - pml_para[ipml*12+3] * Ttmp) * dt * 2.0/(C[posc + 0] + C[posc + NK + 0]);
			 pml_xtxz[pmlpos] = 2.0 * Ttmp - pml_xtxz[pmlpos];
			}
			if(ix > NX - nabs) {
			 // vx
			 ipml = ix - (NX - nabs) - 1;
			 Ttmp = (2.0 * pml_xtxx[pmlpos] + pml_para[ipml*12+4] * xtxx) / (2.0 + pml_para[ipml*12+0]);
			 vx[pos] += ((pml_para[ipml*12+2] - 1) * xtxx / dx - pml_para[ipml*12+2] * Ttmp) * dt * 2.0/(C[posc + 0] + C[posc - NZ*NK + 0]);
			 pml_xtxx[pmlpos] = 2.0 * Ttmp - pml_xtxx[pmlpos];
			}

		// PML at Z direction
			if(iz < fnabs) {
			 ipml = fnabs - 1 - iz;
			 // vx
			 Ttmp = (2.0 * pml_ztxz[pmlpos] + pml_para[ipml*12+10] * ztxz) / (2.0 + pml_para[ipml*12+6]);
			 vx[pos] += ((pml_para[ipml*12+8] - 1) * ztxz / dz - pml_para[ipml*12+8] * Ttmp) * dt * 2.0/(C[posc + 0] + C[posc - NZ*NK + 0]);
			 pml_ztxz[pmlpos] = 2.0 * Ttmp - pml_ztxz[pmlpos];
			 // vz
			 Ttmp = (2.0 * pml_ztzz[pmlpos] + pml_para[ipml*12+11] * ztzz) / (2.0 + pml_para[ipml*12+7]);
			 //vz[pos] += ((pml_beta_half[ipml] - 1) * ztzz / dx - pml_beta_half[ipml] * Ttmp) * dt * 2.0/(C[posc + 0] + C[posc - NZ*NK + 0]);
			 vz[pos] += ((pml_para[ipml*12+9] - 1) * ztzz / dz - pml_para[ipml*12+9] * Ttmp) * dt * 2.0/(C[posc + 0] + C[posc - NZ*NK + 0]);
			 pml_ztzz[pmlpos] = 2.0 * Ttmp - pml_ztzz[pmlpos];
			}

			if(iz >= NZ - nabs) {
			 // vz
			 ipml = iz - (NZ - nabs);
			 Ttmp = (2.0 * pml_ztzz[pmlpos] + pml_para[ipml*12+11] * ztzz) / (2.0 + pml_para[ipml*12+7]);
			 //vz[pos] += ((pml_beta_half[ipml] - 1) * ztzz / dx - pml_beta_half[ipml] * Ttmp) * dt * 2.0/(C[posc + 0] + C[posc - NZ*NK + 0]);
			 vz[pos] += ((pml_para[ipml*12+9] - 1) * ztzz / dz - pml_para[ipml*12+9] * Ttmp) * dt * 2.0/(C[posc + 0] + C[posc - NZ*NK + 0]);
			 pml_ztzz[pmlpos] = 2.0 * Ttmp - pml_ztzz[pmlpos];
			}
			if(iz > NZ - nabs) {
			 // vx
			 ipml = iz - (NZ - nabs) - 1;
			 Ttmp = (2.0 * pml_ztxz[pmlpos] + pml_para[ipml*12+10] * ztxz) / (2.0 + pml_para[ipml*12+6]);
			 vx[pos] += ((pml_para[ipml*12+8] - 1) * ztxz / dz - pml_para[ipml*12+8] * Ttmp) * dt * 2.0/(C[posc + 0] + C[posc - NZ*NK + 0]);
			 pml_ztxz[pmlpos] = 2.0 * Ttmp - pml_ztxz[pmlpos];

			}

      // __syncthreads();
		}

	return;

}

/*
float* get_deviceMem_float(float* ori, int msize){

  float* dev_ori;
  hipError_t err = hipSuccess;
  err = hipMalloc( (void**)&dev_ori, msize * sizeof(float) );
  if (err != hipSuccess) {
    printf("stderr, %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(dev_ori, ori, msize * sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printf("stderr, %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  return dev_ori;
}

int* get_deviceMem_int(int* ori, int msize){

  int* dev_ori;
  hipError_t err = hipSuccess;
  err = hipMalloc( (void**)&dev_ori, msize * sizeof(int) );
  if (err != hipSuccess) {
    printf("stderr, %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(dev_ori, ori, msize * sizeof(int), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printf("stderr, %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  return dev_ori;
}

void copyback_float(float* ori, float* dev_ori, int msize){

  hipError_t err = hipSuccess;
  err = hipMemcpy(ori, dev_ori, msize*sizeof(float), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    printf("stderr, %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

}
	
//void copyback_int(ori, dev_ori, msize){
//
//  err = hipMemcpy(ori, dev_ori, msize*sizeof(int), hipMemcpyDeviceToHost);
//  if (err != hipSuccess) {
//    printf("stderr, %s\n", hipGetErrorString(err));
//    exit(EXIT_FAILURE);
//  }
//}

	
void gpu_update_velocity(float* vx, float* vz, float* txx, float* tzz, float* txz,\
	float* C, float* pml_vxx, float* pml_vzz, float* pml_vxz, float* pml_vzx,\
	float* pml_xtxx, float* pml_xtxz, float* pml_ztxz, float* pml_ztzz, float* pml_para,\
	int NZ, int NX, int NK, int NR, int nabs, int fnabs,\
  int* rec_loc, float* record_vz, float* record_vx, float* div, float* curl,\
  float wave, int* src_loc,\
  int it, float dx, float dz, float dt, int pml_size){

  
  //printf("in host\n");
  hipSetDevice(6);
  //hipError_t err = hipSuccess;
  int ir, pos;
  // local parameters
  float *dev_vx = get_deviceMem_float(vx, NX*NZ);
  float *dev_vz = get_deviceMem_float(vz, NX*NZ);
  float *dev_txx = get_deviceMem_float(txx, NX*NZ);
  float *dev_tzz = get_deviceMem_float(tzz, NX*NZ);
  float *dev_txz = get_deviceMem_float(txz, NX*NZ);
  float *dev_C = get_deviceMem_float(C, NX*NZ*NK);
  float *dev_pml_vxx = get_deviceMem_float(pml_vxx, pml_size);
  float *dev_pml_vzz = get_deviceMem_float(pml_vzz, pml_size);
  float *dev_pml_vxz = get_deviceMem_float(pml_vxz, pml_size);
  float *dev_pml_vzx = get_deviceMem_float(pml_vzx, pml_size);
  float *dev_pml_xtxx = get_deviceMem_float(pml_xtxx, pml_size);
  float *dev_pml_xtxz = get_deviceMem_float(pml_xtxz, pml_size);
  float *dev_pml_ztxz = get_deviceMem_float(pml_ztxz, pml_size);
  float *dev_pml_ztzz = get_deviceMem_float(pml_ztzz, pml_size);
  float *dev_pml_para = get_deviceMem_float(pml_para, pml_size);
  int *dev_rec_loc = get_deviceMem_int(rec_loc, NR);
  int *dev_src_loc = get_deviceMem_int(src_loc, 2);
  

  int block_size = 20;
  dim3 Threads(block_size, block_size, 1);
  dim3 Grids((NX + block_size - 1) / block_size, (NZ + block_size - 1) / block_size, 1);
  // cal
  //printf("in host dev_vx[100] = %f\n", dev_vx[100]);
 // kernel_test <<< 1,1 >>> (dev_vx);
  //kernel_update_velocity <<< Grids, Threads >>> ( dev_vx,  dev_vz,  dev_txx,  dev_tzz,  dev_txz,
  kernel_update_velocity <<< 1,  >>> ( dev_vx,  dev_vz,  dev_txx,  dev_tzz,  dev_txz,\
    dev_C, dev_pml_vxx,  dev_pml_vzz,  dev_pml_vxz,  dev_pml_vzx,\
	  dev_pml_xtxx,  dev_pml_xtxz,  dev_pml_ztxz,  dev_pml_ztzz,  dev_pml_para,\
    dev_rec_loc,  dev_src_loc,  wave,\
	  NZ,  NX,  NK,  NR,  nabs,  fnabs,\
	  dx,  dz,  dt);
  //printf("in host dev_vx[100] = %f\n", dev_vx[100]);

  // copy device parameters to host
  copyback_float(vx, dev_vx, NX*NZ);
  copyback_float(vz, dev_vz, NX*NZ);
  copyback_float(txx, dev_txx, NX*NZ);
  copyback_float(txz, dev_txz, NX*NZ);
  copyback_float(tzz, dev_tzz, NX*NZ);
  //copyback_float(C, dev_vx, NX*NZ*NK);
  copyback_float(pml_vxx, dev_pml_vxx, pml_size);
  copyback_float(pml_vzz, dev_pml_vzz, pml_size);
  copyback_float(pml_vxz, dev_pml_vxz, pml_size);
  copyback_float(pml_vzx, dev_pml_vzx, pml_size);
  copyback_float(pml_xtxx, dev_pml_xtxx, pml_size);
  copyback_float(pml_xtxz, dev_pml_xtxz, pml_size);
  copyback_float(pml_ztxz, dev_pml_ztzz, pml_size);
  copyback_float(pml_ztzz, dev_pml_ztzz, pml_size);
//  copyback_float(pml_para, dev_pml_para, pml_size);

//  printf("in host vx[100] = %f\n", vx[100]);

	// calculate the output data
	for(ir = 0; ir < NR; ir++){
		pos = (rec_loc[ir * 2 + 0] + nabs) * NZ + rec_loc[ir * 2 + 1] + fnabs;
//		record_vx[it * NR + ir] = vx[pos];
//		record_vz[it * NR + ir] = vz[pos];
		record_vx[it * NR + ir] = 0.5 * ( vx[pos] + vx[pos + NZ] );
		record_vz[it * NR + ir] = 0.5 * ( vz[pos] + vz[pos - 1] );
		div[it * NR + ir] =  (coe1 * (vx[pos + NZ] - vx[pos])
			 			    + coe2 * (vx[pos + 2*NZ] - vx[pos - NZ])) / dx
			 				+(coe1 * (vz[pos] - vz[pos - 1])
			 				+ coe2 * (vz[pos + 1] - vz[pos - 2])) / dz;
		curl[it * NR + ir] = 0.25 * (
			( (coe1*(vx[pos] - vx[pos-1])+coe2*(vx[pos+1] - vx[pos-2]))/dz) 
			- ( (coe1*(vz[pos-1] - vz[pos-NZ-1])+coe2*(vz[pos+NZ-1] - vz[pos-2*NZ-1]))/dx)
			+( (coe1*(vx[pos+1] - vx[pos])+coe2*(vx[pos+2] - vx[pos-1]))/dz) 
			- ( (coe1*(vz[pos] - vz[pos-NZ])+coe2*(vz[pos+NZ] - vz[pos-2*NZ]))/dx)
			+( (coe1*(vx[pos+NZ] - vx[pos+NZ-1])+coe2*(vx[pos+NZ+1] - vx[pos+NZ-2]))/dz) 
			- ( (coe1*(vz[pos+NZ-1] - vz[pos-1])+coe2*(vz[pos+2*NZ-1] - vz[pos-NZ-1]))/dx)
			+( (coe1*(vx[pos+NZ+1] - vx[pos+NZ])+coe2*(vx[pos+NZ+2] - vx[pos+NZ-1]))/dz) 
			- ( (coe1*(vz[pos+NZ] - vz[pos])+coe2*(vz[pos+2*NZ] - vz[pos-NZ]))/dx)
			);
  }

//  free(dev_vx);
  hipFree(dev_vx);
  hipFree(dev_vz);
  hipFree(dev_txx);
  hipFree(dev_txz);
  hipFree(dev_tzz);
  hipFree(dev_C);
  hipFree(dev_pml_vxx);
  hipFree(dev_pml_vzz);
  hipFree(dev_pml_vxz);
  hipFree(dev_pml_vzx);
  hipFree(dev_pml_xtxx);
  hipFree(dev_pml_xtxz);
  hipFree(dev_pml_ztxz);
  hipFree(dev_pml_ztzz);
  hipFree(dev_pml_para);
  hipFree(dev_rec_loc);
  hipFree(dev_src_loc);

  }
*/
	
