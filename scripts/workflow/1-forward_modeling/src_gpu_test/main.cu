#include "hip/hip_runtime.h"
/**************************************
***		Forward modeling			***
*** 	Author: Yanwen Wei          ***
*** 	Date: 2019-9-23             ***
***		Email: wei_yanwen@163.com   ***
***************************************/


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "common.h"
#include "kernel.cuh"


float* get_deviceMem_float(float* ori, int msize){

  float* dev_ori;
  hipError_t err = hipSuccess;
  err = hipMalloc( (void**)&dev_ori, msize * sizeof(float) );
  if (err != hipSuccess) {
    printf("stderr, %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(dev_ori, ori, msize * sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printf("stderr, %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  return dev_ori;
}

int* get_deviceMem_int(int* ori, int msize){

  int* dev_ori;
  hipError_t err = hipSuccess;
  err = hipMalloc( (void**)&dev_ori, msize * sizeof(int) );
  if (err != hipSuccess) {
    printf("stderr, %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(dev_ori, ori, msize * sizeof(int), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printf("stderr, %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  return dev_ori;
}

void copyback_float(float* ori, float* dev_ori, int msize){

  hipError_t err = hipSuccess;
  err = hipMemcpy(ori, dev_ori, msize*sizeof(float), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    printf("stderr, %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

}

void copy_float(float* ori, float* dev_ori, int msize){

  hipError_t err = hipSuccess;
  err = hipMemcpy(dev_ori, ori, msize*sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printf("stderr, %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

}
void copy_int(int* ori, int* dev_ori, int msize){

  hipError_t err = hipSuccess;
  err = hipMemcpy(dev_ori, ori, msize*sizeof(int), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printf("stderr, %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}






int main(int argc, char** argv){

// parameters
	time_t t0;
	clock_t start, stop;
  int readin_model, save_snap, readin_source, readin_receiver;
  int snap_step;
	int is, ir, it, ix, iz, nabs, fnabs, pos;
	int nx, nz, NX, NZ, NK, NS, NR, NT;
	int sz, sx0, sdx, rz0, rdz, rx;
	int *src_loc, *rec_loc, *this_src_loc;
	float dx, dz, dt, freq0, amp0;
	float Vmin, Vmax, wave;
	float *record_vz, *record_vx, *div, *curl, *snap_vz, *snap_vx;
	float *C, *txx, *tzz, *txz, *vx, *vz;// readin C
	float *pml_vxx, *pml_vzz, *pml_vzx, *pml_vxz;//NX*NZ - nx*nnz
	float *pml_xtxx, *pml_xtxz, *pml_ztxz, *pml_ztzz;//NX*NZ - nx*nnz
  float *pml_para;// nabs*12
	char logfile[25], paramfile[25], modelfile[25];
  char sourcefile[25], receiverfile[25], snapf_vz[126], snapf_vx[126];
	char outfile_vz[80], outfile_vx[80], outfile_div[80], outfile_curl[80];
	FILE *fid, *pafid, *mdfid, *oufid, *srfid;


	// default value of the parameters
	// ****************************************************************
  readin_model = 0; save_snap = 0; 
  readin_source = 0; readin_receiver = 0;
	sprintf(logfile,"log_m%s_s%s.txt", argv[1], argv[2]);
//	sprintf(paramfile, "ParamInput.txt");
  sprintf(paramfile, "ParamInput_%s.txt", argv[2]);
	//sprintf(modelfile, "modelfile.bin");
	// sprintf(outfile_vz, "vz.bin");
	// sprintf(outfile_vx, "vx.bin");
	// sprintf(outfile_div, "div.bin");
	// sprintf(outfile_curl, "curl.bin");
	//sprintf(sourcefile, "sourcefile.txt");
	//sprintf(receiverfile, "receiverfile.txt");
	nx = 300; nz = 400;
	nabs = 40; fnabs = 40;
	NX = nx + 2 * nabs; NZ = nz + fnabs + nabs;
	NS = 1; NR = 0; NT = 100; NK = 4;
	dx = 5; dz = 5; dt = 0.001; freq0 = 20; amp0 = 1;
  snap_step = int(0.1 / dt);
	// ****************************************************************

	time(&t0);//local time


// prepare the log file
	// sprintf(logfile,"log.txt");
	if( !(fid=fopen(logfile,"w")) ) 
		{ printf("log file not opened"); return(1); }
	fprintf(fid, "***************************************************\n");
	fprintf(fid, "\tBegin to Produce Synthetic Seismic Data \n");
	fprintf(fid, "\tstart time: %s",ctime(&t0));
	fprintf(fid, "***************************************************\n\n");

// readin parameters
	if( !(pafid=fopen(paramfile,"r")) ) {
		fprintf(fid,"Error opening param file %s\n",paramfile);}
	else{
		fprintf(fid,"Start reading parameters in file %s\n", paramfile);
		fscanf(pafid,"%i %i %f %f", &nx, &nz, &dx, &dz);
		fscanf(pafid,"%i %i %i %i", &sz, &sx0, &sdx, &NS);
		fscanf(pafid,"%i %i %i %i", &rx, &rz0, &rdz, &NR);
		fscanf(pafid,"%f %f %f %i", &freq0, &amp0, &dt, &NT);
		fscanf(pafid,"%i %i", &nabs, &fnabs);
		fscanf(pafid,"%i", &readin_model);
		fscanf(pafid,"%i %i", &save_snap, &snap_step);
		fscanf(pafid,"%i %i", &readin_source, &readin_receiver);
	}
	fclose(pafid);
// update parameters and initial matrix
	fprintf(fid, "***\t\tThe running parameters\t\t***\n");
  fprintf(fid, "nx %i, nz %i dx %f dz %f\n", nx, nz, dx, dz);
  fprintf(fid, "sz %i, sx0 %i sdx %i NS %i\n", sz, sx0, sdx, NS);
  fprintf(fid, "rx %i, rz0 %i rdz %i NR %i\n", rx, rz0, rdz, NR);
  fprintf(fid, "freq0 %f, amp0 %f dt %f NT %i\n", freq0, amp0, dt, NT);
  fprintf(fid, "nabs %i, fnabs %i\n", nabs, fnabs);
  fprintf(fid, "save_snap %i, snap_step %i\n", save_snap, snap_step);
  fprintf(fid, "readin_source %i, readin_receiver %i\n", readin_source, readin_receiver);
	// C = init_matrix( NX * NZ * NK);


// init all the global matrix
	NX = nx + 2 * nabs; NZ = nz + fnabs + nabs;
	C = init_matrix( NX * NZ * NK);
	vx = init_matrix( NX * NZ );
	vz = init_matrix( NX * NZ );
	txx = init_matrix( NX * NZ );
	tzz = init_matrix( NX * NZ );
	txz = init_matrix( NX * NZ );

	pml_vxx = init_matrix( NX * NZ - nx * nz);
	pml_vzz = init_matrix( NX * NZ - nx * nz);
	pml_vxz = init_matrix( NX * NZ - nx * nz);
	pml_vzx = init_matrix( NX * NZ - nx * nz);
	pml_xtxx = init_matrix( NX * NZ - nx * nz);
	pml_xtxz = init_matrix( NX * NZ - nx * nz);
	pml_ztxz = init_matrix( NX * NZ - nx * nz);
	pml_ztzz = init_matrix( NX * NZ - nx * nz);

  pml_para = init_matrix( nabs*12 );// 6 * 2

	src_loc = init_matrix_int( 2 * NS);
	rec_loc = init_matrix_int( 2 * NR);
	this_src_loc = init_matrix_int( 2 );
	record_vz = init_matrix( NR * NT );
	record_vx = init_matrix( NR * NT );
	div = init_matrix( NR * NT );
	curl = init_matrix( NR * NT );
	snap_vz = init_matrix( nx * nz );
	snap_vx = init_matrix( nx * nz );


  // set the source and receiver locations
  if (! readin_source){
	  for(is = 0; is < NS; is++){
	  	src_loc[2 * is + 0] = sx0 + sdx * is;
	  	src_loc[2 * is + 1] = sz;
	  }
  }
  else{
    sprintf(sourcefile, "sourcefile.txt");
    if( !(srfid=fopen(sourcefile,"r")) ) {
    fprintf(fid,"Error opening param file %s\n", sourcefile);
    	return(1); }
    else{
     fprintf(fid,"\n\nStart reading sourcefile %s\n", sourcefile);
     readin_coor(srfid, NS, src_loc, dx, dz);
    } 
  }
  if (! readin_receiver){
  	for(ir = 0; ir < NR; ir++){
  		rec_loc[2 * ir + 0] = rx;
  		rec_loc[2 * ir + 1] = rz0 + rdz * ir;
  	}
  }
  else{
    sprintf(receiverfile, "receiverfile.txt");
    if( !(srfid=fopen(receiverfile,"r")) ) {
    fprintf(fid,"Error opening param file %s\n", receiverfile);
    	return(1); }
    else{
     fprintf(fid,"\n\nStart reading receiverfile %s\n", receiverfile);
     readin_coor(srfid, NR, rec_loc, dx, dz);
    }
  }

  fprintf(fid,"\n\n souce position \n\n");
  for(ix=0; ix<NS; ix++){fprintf(fid, "%i %i\n", src_loc[ix*2+0], src_loc[ix*2+1]);}
  fprintf(fid,"\n\n receiver position \n\n");
  for(ix=0; ix<NR; ix++){fprintf(fid, "%i %i\n", rec_loc[ix*2+0], rec_loc[ix*2+1]);}

 // readin model and initial C
 sprintf(modelfile, "modelfile_%s.bin", argv[1]);
 if( !(mdfid=fopen(modelfile,"rb")) ) {
 fprintf(fid,"Error opening param file %s\n", modelfile);
 	return(1); }
 else{
  fprintf(fid,"\n\nStart reading modelfile %s\n", modelfile);
  Vmax = 0.; Vmin = 1e10;
  get_C (mdfid, NK, NZ, NX, nabs, fnabs, readin_model, C, Vmax, Vmin);
 }

// check
  fprintf(fid, "***CHECKPOINT***\n");
  fprintf(fid, "Vmax, %f\n", Vmax);
  fprintf(fid, "Vmix, %f\n", Vmin);
  fprintf(fid, "VZ, %f\n", vz[100]);

  set_pml(nabs, dx, dz, dt, Vmax, freq0, pml_para);



  hipSetDevice(atoi(argv[3]));

  int pml_size = NZ*NX - nz*nx;
  float *dev_vx = get_deviceMem_float(vx, NX*NZ);
  float *dev_vz = get_deviceMem_float(vz, NX*NZ);
  float *dev_txx = get_deviceMem_float(txx, NX*NZ);
  float *dev_tzz = get_deviceMem_float(tzz, NX*NZ);
  float *dev_txz = get_deviceMem_float(txz, NX*NZ);
  float *dev_C = get_deviceMem_float(C, NX*NZ*NK);
  float *dev_pml_vxx = get_deviceMem_float(pml_vxx, pml_size);
  float *dev_pml_vzz = get_deviceMem_float(pml_vzz, pml_size);
  float *dev_pml_vxz = get_deviceMem_float(pml_vxz, pml_size);
  float *dev_pml_vzx = get_deviceMem_float(pml_vzx, pml_size);
  float *dev_pml_xtxx = get_deviceMem_float(pml_xtxx, pml_size);
  float *dev_pml_xtxz = get_deviceMem_float(pml_xtxz, pml_size);
  float *dev_pml_ztxz = get_deviceMem_float(pml_ztxz, pml_size);
  float *dev_pml_ztzz = get_deviceMem_float(pml_ztzz, pml_size);
  float *dev_pml_para = get_deviceMem_float(pml_para, nabs*12);
  int *dev_rec_loc = get_deviceMem_int(rec_loc, NR);
  int *dev_src_loc = get_deviceMem_int(src_loc, 2);
  int block_size = 32;
  dim3 Threads(block_size, block_size, 1);
  //dim3 Grids((NX + block_size - 1) / block_size, (NZ + block_size - 1) / block_size, 1);
  dim3 Grids((NX - 1) / block_size + 1 , (NZ - 1) / block_size + 1, 1);

// kernel 
  time(&t0);//local time
  fprintf(fid, "\n\n***************************************************\n");
  fprintf(fid, "\t loops start at time %s", ctime(&t0));
  fprintf(fid, "***************************************************");

  for (is = 0; is < NS; is++){

  	this_src_loc[0] = src_loc[is*2];
  	this_src_loc[1] = src_loc[is*2 + 1];
    copy_int(this_src_loc, dev_src_loc, 2);
    //int *dev_src_loc = get_deviceMem_int(this_src_loc, 2);
//    checkCudaErrors( hipMemcpy(dev_src_loc, this_src_loc, 2*sizeof(int), hipMemcpyHostToDevice) );
    hipDeviceSynchronize();
  	start = clock();

  	for (it = 0; it < NT; it++){

  		wave = Ricker(it*dt, freq0, amp0);
      wave = - wave * dt / (dx * dx);
  //		printf("it %d, wave %f\n", it, wave);
      
  		///update_stress(vx, vz, txx, tzz, txz,\
			  C, pml_vxx, pml_vzz, pml_vxz, pml_vzx, pml_para,\
			  NZ, NX, NK, nabs, fnabs,\
			  wave, this_src_loc, dx, dz, dt);
      kernel_update_stress <<< Grids, Threads >>> ( dev_vx,  dev_vz,  dev_txx,  dev_tzz,  dev_txz,\
        dev_C, dev_pml_vxx,  dev_pml_vzz,  dev_pml_vxz,  dev_pml_vzx, dev_pml_para,\
	      NZ,  NX,  NK,  nabs,  fnabs,\
	      wave, dev_src_loc, dx,  dz,  dt);
      hipDeviceSynchronize();
     // copyback_float(txx, dev_txx, NX*NZ);
     // copyback_float(tzz, dev_tzz, NX*NZ);
     // copyback_float(txz, dev_txz, NX*NZ);
     // hipDeviceSynchronize();
	    //update_velocity(vx, vz, txx, tzz, txz,\
			  C, pml_vxx, pml_vzz, pml_vxz, pml_vzx,\
			  pml_xtxx, pml_xtxz, pml_ztxz, pml_ztzz, pml_para,\
			  NZ, NX, NK, NR, nabs, fnabs,\
			  rec_loc, record_vz, record_vx, div, curl,\
        wave, this_src_loc, it, dx, dz, dt);
      //copy_float(vx, dev_vx, NX*NZ);
      //copy_float(vz, dev_vz, NX*NZ);

	    //gpu_update_velocity(vx, vz, txx, tzz, txz,\
			  C, pml_vxx, pml_vzz, pml_vxz, pml_vzx,\
			  pml_xtxx, pml_xtxz, pml_ztxz, pml_ztzz, pml_para,\
			  NZ, NX, NK, NR, nabs, fnabs,\
			  rec_loc, record_vz, record_vx, div, curl,\
        wave, this_src_loc, it, dx, dz, dt, NX*NZ-nz*nx);
      kernel_update_velocity <<< Grids, Threads >>> ( dev_vx,  dev_vz,  dev_txx,  dev_tzz,  dev_txz,\
       dev_C,	dev_pml_xtxx,  dev_pml_xtxz,  dev_pml_ztxz,  dev_pml_ztzz,  dev_pml_para,\
       dev_rec_loc,  dev_src_loc,  wave,\
	     NZ,  NX,  NK,  NR,  nabs,  fnabs,\
	     dx,  dz,  dt, it);
/*  
     //kernel_update <<< Grids, Threads >>> ( dev_vx,  dev_vz,  dev_txx,  dev_tzz,  dev_txz,\
       dev_C, dev_pml_vxx,  dev_pml_vzz,  dev_pml_vxz,  dev_pml_vzx,\
	     dev_pml_xtxx,  dev_pml_xtxz,  dev_pml_ztxz,  dev_pml_ztzz,  dev_pml_para,\
       dev_rec_loc,  dev_src_loc,  wave,\
	     NZ,  NX,  NK,  NR,  nabs,  fnabs,\
	     dx,  dz,  dt);
       */
     hipDeviceSynchronize();
     copyback_float(vx, dev_vx, NX*NZ);
     copyback_float(vz, dev_vz, NX*NZ);

   
	  for(ir = 0; ir < NR; ir++){
	  	pos = (rec_loc[ir * 2 + 0] + nabs) * NZ + rec_loc[ir * 2 + 1] + fnabs;
//  		record_vx[it * NR + ir] = vx[pos];
//  		record_vz[it * NR + ir] = vz[pos];
	  	record_vx[it * NR + ir] = 0.5 * ( vx[pos] + vx[pos + NZ] );
	  	record_vz[it * NR + ir] = 0.5 * ( vz[pos] + vz[pos - 1] );
	  	div[it * NR + ir] =  (coe1 * (vx[pos + NZ] - vx[pos])
	  		 			    + coe2 * (vx[pos + 2*NZ] - vx[pos - NZ])) / dx
	  		 				+(coe1 * (vz[pos] - vz[pos - 1])
	  		 				+ coe2 * (vz[pos + 1] - vz[pos - 2])) / dz;
	  	curl[it * NR + ir] = 0.25 * (
	  		( (coe1*(vx[pos] - vx[pos-1])+coe2*(vx[pos+1] - vx[pos-2]))/dz) 
	  		- ( (coe1*(vz[pos-1] - vz[pos-NZ-1])+coe2*(vz[pos+NZ-1] - vz[pos-2*NZ-1]))/dx)
	  		+( (coe1*(vx[pos+1] - vx[pos])+coe2*(vx[pos+2] - vx[pos-1]))/dz) 
	  		- ( (coe1*(vz[pos] - vz[pos-NZ])+coe2*(vz[pos+NZ] - vz[pos-2*NZ]))/dx)
	  		+( (coe1*(vx[pos+NZ] - vx[pos+NZ-1])+coe2*(vx[pos+NZ+1] - vx[pos+NZ-2]))/dz) 
	  		- ( (coe1*(vz[pos+NZ-1] - vz[pos-1])+coe2*(vz[pos+2*NZ-1] - vz[pos-NZ-1]))/dx)
	  		+( (coe1*(vx[pos+NZ+1] - vx[pos+NZ])+coe2*(vx[pos+NZ+2] - vx[pos+NZ-1]))/dz) 
	  		- ( (coe1*(vz[pos+NZ] - vz[pos])+coe2*(vz[pos+2*NZ] - vz[pos-NZ]))/dx)
	  		);
       
    	}

    if (save_snap && it % snap_step == 0){
      for (ix=0; ix < nx; ix++)
        for (iz=0; iz < nz; iz++){
          snap_vz[ix*nz + iz] = vz[(ix+nabs)*NZ + iz + fnabs];
          snap_vx[ix*nz + iz] = vx[(ix+nabs)*NZ + iz + fnabs];
      }
  	  sprintf(snapf_vz, "snap_output/model_%s_shot_%s_it_%d_vz.bin", argv[1], argv[2], it);
  	  if( !(oufid=fopen(snapf_vz,"wb")) ) {
		  fprintf(fid,"Error opening output file %s\n", snapf_vz);
      	return(1); }
  	  else fwrite(snap_vz, sizeof(float), nz*nx, oufid);
  	  fclose(oufid);
  	  sprintf(snapf_vx, "snap_output/model_%s_shot_%s_it_%d_vx.bin", argv[1], argv[2], it);
  	  if( !(oufid=fopen(snapf_vx,"wb")) ) {
		  fprintf(fid,"Error opening output file %s\n", snapf_vx);
      	return(1); }
  	  else fwrite(snap_vx, sizeof(float), nz*nx, oufid);
  	  fclose(oufid);
      }
  
  

       
    }// end of time iteration

  	stop = clock();
  	fprintf(fid, "\nFinish shot %d, time consuming %f sec.", is, (double)(stop - start)/CLOCKS_PER_SEC);




  	sprintf(outfile_vz, "output/model_%s_shot_%s_vz.bin", argv[1], argv[2]);
  	sprintf(outfile_vx, "output/model_%s_shot_%s_vx.bin", argv[1], argv[2]);
  	sprintf(outfile_div, "output/model_%s_shot_%s_div.bin", argv[1], argv[2]);
  	sprintf(outfile_curl, "output/model_%s_shot_%s_curl.bin", argv[1], argv[2]);

  	if( !(oufid=fopen(outfile_vz,"wb")) ) {
		fprintf(fid,"Error opening output file %s\n", outfile_vz);
    	return(1); }
  	else fwrite(record_vz, sizeof(float), NT*NR, oufid);
  	fclose(oufid);
  
  	if( !(oufid=fopen(outfile_vx,"wb")) ) {
  		fprintf(fid,"Error opening output file %s\n", outfile_vx);
      	return(1); }
  	else fwrite(record_vx, sizeof(float), NT*NR, oufid);
  	fclose(oufid);
  
  	if( !(oufid=fopen(outfile_div,"wb")) ) {
  		fprintf(fid,"Error opening output file %s\n", outfile_div);
      	return(1); }
  	else fwrite(div, sizeof(float), NT*NR, oufid);
  	fclose(oufid);
  
  	if( !(oufid=fopen(outfile_curl,"wb")) ) {
  		fprintf(fid,"Error opening output file %s\n", outfile_curl);
      	return(1); }
  	else fwrite(curl, sizeof(float), NT*NR, oufid);
  	fclose(oufid);

  }


  hipFree(dev_vx);
  hipFree(dev_vz);
  hipFree(dev_txx);
  hipFree(dev_txz);
  hipFree(dev_tzz);
  hipFree(dev_C);
  hipFree(dev_pml_vxx);
  hipFree(dev_pml_vzz);
  hipFree(dev_pml_vxz);
  hipFree(dev_pml_vzx);
  hipFree(dev_pml_xtxx);
  hipFree(dev_pml_xtxz);
  hipFree(dev_pml_ztxz);
  hipFree(dev_pml_ztzz);
  hipFree(dev_pml_para);
  hipFree(dev_rec_loc);
  hipFree(dev_src_loc);



// close
	free(vx);free(vz);free(txx);free(tzz);free(txz);
	free(pml_vxx);free(pml_vxz);free(pml_vzz);free(pml_vzx);
	free(pml_xtxx);free(pml_ztzz);free(pml_xtxz);free(pml_ztxz);
  free(pml_para);
	free(C);free(rec_loc);free(src_loc);free(this_src_loc);
	free(record_vz);free(record_vx);free(div);free(curl);
  free(snap_vz); free(snap_vx);
	time(&t0);//local time
    fprintf(fid, "\n\n***************************************************\n");
    fprintf(fid, "\t done at time %s", ctime(&t0));
    fprintf(fid, "***************************************************");
	fclose(fid);
	
	return 0;
}
