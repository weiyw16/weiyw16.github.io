#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"


float* init_matrix(int sizenum){
	int i;
	float *U;
	if( ! (U = (float *)malloc(sizenum*sizeof(float))) )
      printf("init failed !\n");
	for (i = 0; i < sizenum; i++) U[i] = 0;
	return U;
}

int* init_matrix_int(int sizenum){
	int i;
	int *U;
	U = (int *)malloc((sizenum*sizeof(int)));
	for (i = 0; i < sizenum; i++) U[i] = 0;
	return U;
}

float Ricker(float time, float freq0, float amp0 ){
	float wave, a;
	time = time - ( 1.5/freq0 ); // delay
	//time = time - ( 1.5/freq0 ); // delay
  a = freq0 * sqrt(2.) * pi;
  // if (time > tsrcf) wave = 0.;
  wave = (1. - a*a*time*time)*exp(-0.5*a*a*time*time);// Ricker
  //wave = a*a*time*(a*a*time*time-3.0)*exp(-0.5*a*a*time*time);  // Ricker derivative
  wave = wave * amp0;
  return(wave);
}


void set_pml(int nabs, float dx, float dz, float dt, float Vmax, float freq0, float* pml_para){

	int i, id;
	float lnR, thk_pml, beta0, alpha0, d0, xL;
	float d0factor = 3.0;
	float PPW0=10.0;
	float p_power=2.0;

	// natural logarithm of the theoretical reflection coefficient R
	lnR = log(10) * (-3.0 - (log10( nabs + 0.0 ) - 1.0) / log10(2.0));
	// thickness of PML

  for(id = 0; id < 2; id++) {
    if(id == 0) dx = dx;
    if(id == 1) dx = dz;
    //printf("dx %f\n", dx);
	  thk_pml = nabs * dx;
	  d0 = -(p_power + 1.0) * Vmax * lnR / ( 2.0 * thk_pml);
	  d0 = d0 * d0factor;
	  beta0 = Vmax / (0.5 * PPW0 * dx * freq0);
	  if(beta0 < 1.0)  beta0 = 1.0;
	  alpha0 = pi * freq0;

	  for(i = 0; i < nabs; i++) {
	  	// i=0: interior interface; i=nabs-1: exterior boundry;

	  	// define damping profile at grid points
	  	xL=(i+1)*dx/thk_pml;
	  	pml_para[i*12+id*6+4]=d0*pow(xL,p_power);
	  	pml_para[i*12+id*6+2]=1.0+(beta0-1.0)*pow(xL,p_power);
	  	pml_para[i*12+id*6+0]=alpha0*(1.0-xL);

	  	// define damping profile at half grid points
	  	//xL=(i+1)*dx/thk_pml;
	  	xL=(i+0.5)*dx/thk_pml;
	  	pml_para[i*12+id*6+5]=d0*pow(xL,p_power);
	  	pml_para[i*12+id*6+3]=1.0+(beta0-1.0)*pow(xL,p_power);
	    pml_para[i*12+id*6+1]=alpha0*(1.0-xL);

	    if(pml_para[i*12+id*6+0]<0.0)  pml_para[i*12+id*6+0]=0.0;
	  	if(pml_para[i*12+id*6+1]<0.0)  pml_para[i*12+id*6+1]=0.0;

	  	// beta <-- 1/beta
	  	pml_para[i*12+id*6+2]=1.0/pml_para[i*12+id*6+2];
	  	pml_para[i*12+id*6+3]=1.0/pml_para[i*12+id*6+3];

	  	// d <-- d/beta
	  	pml_para[i*12+id*6+4]=pml_para[i*12+id*6+4]*pml_para[i*12+id*6+2];
	  	pml_para[i*12+id*6+5]=pml_para[i*12+id*6+5]*pml_para[i*12+id*6+3];

	  	// alpha <-- alpha + d/beta
	  	pml_para[i*12+id*6+0]=pml_para[i*12+id*6+0]+pml_para[i*12+id*6+4];
	  	pml_para[i*12+id*6+1]=pml_para[i*12+id*6+1]+pml_para[i*12+id*6+5];

	  	// multiply alpha+d/beta by dt
	  	pml_para[i*12+id*6+0]=dt*pml_para[i*12+id*6+0];
	  	pml_para[i*12+id*6+1]=dt*pml_para[i*12+id*6+1];
	  	// multiply d/beta by dt/dx
	  	pml_para[i*12+id*6+4]=dt/dx*pml_para[i*12+id*6+4];
	  	pml_para[i*12+id*6+5]=dt/dx*pml_para[i*12+id*6+5];
    }
  }

  return;
}

void get_C(FILE* mdfid, int NK, int NZ, int NX, int nabs, int fnabs, int readin_model,\
    float* C, float& Vmax, float& Vmin){

  //float Vmax = 0.;
  //float Vmin = 1e10;
  float rho = 2.200;
  float vptmp = 2000;
  float vstmp = 1200;
  int ix, iz, ik;
  float s1, s2, s3, tmpf;
  //for( ix = nabs; ix < NX - nabs; ix++ )
    for( iz = fnabs; iz < NZ - nabs; iz++ )
      for( ix = nabs; ix < NX - nabs; ix++ ) {
      if(readin_model){
   		  fread(&tmpf,sizeof(float),1,mdfid); s1=tmpf;//vp
   		  fread(&tmpf,sizeof(float),1,mdfid); s2=tmpf;//vs
   		  fread(&tmpf,sizeof(float),1,mdfid); s3=tmpf*0.001;//rho
        C[ix * NK * NZ + iz * NK + 0] = s3;//rho;//s3;
        C[ix * NK * NZ + iz * NK + 1] = s3*s1*s1;//rho*vptmp*vptmp*dt/dx;//lam2mu;
        C[ix * NK * NZ + iz * NK + 2] = s3*s1*s1 - 2*s3*s2*s2;//;//lam;
        C[ix * NK * NZ + iz * NK + 3] = s3*s2*s2;//rho*vstmp*vstmp*dt/dx;//mu;
        if (s1 > Vmax) Vmax = s1;
        if (s2 < Vmin) Vmin = s2;
      }
      else{
        C[ix * NK * NZ + iz * NK + 0] = rho;//s3;
        C[ix * NK * NZ + iz * NK + 1] = rho*vptmp*vptmp;//lam2mu;
        C[ix * NK * NZ + iz * NK + 2] = rho*vptmp*vptmp - 2*rho*vstmp*vstmp;//lam;
        C[ix * NK * NZ + iz * NK + 3] = rho*vstmp*vstmp;//mu;
        if (s1 > Vmax) Vmax = vptmp;//s1;
        if (s2 < Vmin) Vmin = vstmp;// s2;

      }
   	}
	fclose(mdfid);
	//printf("%f\n", C[nabs*NZ*4 + fnabs*4 + 0]);
    //boundary
  for(ix = 0; ix < nabs; ix++)
  	for(iz = fnabs; iz < NZ - nabs; iz++)
  		for(ik = 0; ik < 4; ik++ )
  			C[ix * NK * NZ + iz * NK + ik] = C[nabs * NK * NZ + iz * NK + ik];
  for(ix = NX - nabs; ix < NX; ix++ )
   	for(iz = fnabs; iz < NZ - nabs; iz++ )
   		for(ik = 0; ik < 4; ik++ )
   			C[ix * NK * NZ + iz * NK + ik] = C[(NX - nabs - 1) * NK * NZ + iz * NK + ik];
  for(ix = 0; ix < NX; ix++ )
   	for(iz = 0; iz < fnabs; iz++ )
   		for(ik = 0; ik < 4; ik++ )
   			C[ix * NK * NZ + iz * NK + ik] = C[ix * NK * NZ + fnabs * NK + ik];
  for(ix = 0; ix < NX; ix++ )
   	for(iz = NZ - nabs; iz < NZ; iz++ )
   		for(ik = 0; ik < 4; ik++ )
   			C[ix * NK * NZ + iz * NK + ik] = C[ix * NK * NZ + (NZ - nabs - 1) * NK + ik];

}


void readin_coor(FILE* srfid, int N, int* loc, float dx, float dz){

  int ii;
  float tmpx, tmpz;
  for (ii=0; ii < N; ii++){
    fscanf(srfid, "%f %f",&tmpx, &tmpz);
    loc[ii*2+0] = int(tmpx / dx );
    loc[ii*2+1] = int(tmpz / dz );
  }

  fclose(srfid);
  return;
}

void mymemorycp(float* buf_array, float* tar, int start, int len){

	int ii;
	for (ii = 0; ii < len; ii++){
		buf_array[ii] =  tar[start + ii];
	}
	return;
}
