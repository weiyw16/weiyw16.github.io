#include "hip/hip_runtime.h"
#include <stdio.h>
#include "common.h"

/*
float get_cc(float av, float bv, float cv, float dv){

  float cc = 0;
  float erro = 0.0000000001;
  if ( av <= erro || bv <= erro || cv <= erro || dv <= erro )
    cc = 0.;
  else {  //c55=4.0/(1.0/av+1.0/bv+1.0/c+1.0/dv)
    cc = (4.0*av*bv*cv*dv ) / (bv*cv*dv + av*cv*dv + av*bv*dv + av*bv*cv);
  }
  return cc;
}
*/
__global__ void kernel_update_stress (float* vx, float* vz, float* txx, float* tzz, float* txz,\
	float* C, float* pml_vxx, float* pml_vzz, float* pml_vxz, float* pml_vzx, float* pml_para,\
	int NZ, int NX, int NK, int nabs, int fnabs,\
	float wave, int* src_loc, float dx, float dz, float dt){

	// printf("load in stress update\n");
	//local parameters
	int ix, iz, pos, posc, ipml, zcom, xcom, pmlpos;
	float Ctmp, Vtmp;
	float Dvxdz, Dvxdx, Dvzdz, Dvzdx;
  float dtdx, dtdz;

  dtdx = dt/dx;
  dtdz = dt/dz;

  //__syncthreads();

  ix = blockIdx.x * blockDim.x + threadIdx.x;
  iz = blockIdx.y * blockDim.y + threadIdx.y;

	//add source
  if (ix == ( src_loc[0] + nabs ) && iz == ( src_loc[1] + fnabs ) ){
	  txx[ (src_loc[0] + nabs) * NZ + src_loc[1] + fnabs] += wave;
	  tzz[ (src_loc[0] + nabs) * NZ + src_loc[1] + fnabs] += wave;
  }
  //printf("%d\n", blockIdx.x);
	// calculate stress
  if(ix > 0 &&  ix < NX - 1 && iz > 0 && iz < NZ - 1){
//	for (ix = 1; ix < NX - 1; ix++)
//		for (iz = 1; iz < NZ - 1; iz++){
			pos = ix * NZ + iz;
			posc = ix * NZ * NK + iz * NK;


			if ( ix == 1 || ix == NX - 2 ) {
			   Dvxdx = vx[pos+NZ] - vx[pos];//m!=mm-1
			   Dvzdx = vz[pos] - vz[pos-NZ];//m!=0
			}
			else {
			   Dvxdx = coe1*(vx[pos+NZ]-vx[pos]) + coe2*(vx[pos+2*NZ]-vx[pos-NZ]);  /* for Txx and Tzz, at (m+1/2,k) */
			   Dvzdx = coe1*(vz[pos]-vz[pos-NZ]) + coe2*(vz[pos+NZ]-vz[pos-2*NZ]);  /* for Txz, at (m,k+1/2) */
			}

			if( iz == 1 || iz == NZ - 2 ) {
			   Dvzdz = vz[pos] - vz[pos-1];//k!=0
			   Dvxdz = vx[pos+1] - vx[pos];//k!=kk-1
			}
			else {
			   Dvzdz = coe1*(vz[pos]-vz[pos-1]) + coe2*(vz[pos+1]-vz[pos-2]);   /* for Txx and Tzz, at (m+1/2,k) */
			   Dvxdz = coe1*(vx[pos+1]-vx[pos]) + coe2*(vx[pos+2]-vx[pos-1]);   /* for Txz, at (m,k+1/2) */
			}



//      Ctmp = get_cc( C[posc + 3], C[posc + 3 - NZ*NK], C[posc + NK + 3], C[posc + NK + 3 - NZ*NK]);
      if( C[posc + 3] <= erro || C[posc + 3 - NZ*NK] <= erro || C[posc + NK + 3] <= erro || C[posc + NK + 3 - NZ*NK] <= erro )
        Ctmp = 0;
      else {
        //Ctmp = 4.0 / ( 1.0 / C[posc + 3] + 1.0 / C[posc + 3 - NZ*NK] + 1.0 / C[posc + NK + 3] + 1.0 / C[posc + NK + 3 - NZ*NK])
        Ctmp = ( 4.0 * C[posc + 3] * C[posc + 3 - NZ*NK] * C[posc + NK + 3] * C[posc + NK + 3 - NZ*NK] ) \
               / ( C[posc + 3 - NZ*NK] * C[posc + NK + 3] * C[posc + NK + 3 - NZ*NK] \
                   + C[posc + 3] * C[posc + NK + 3] * C[posc + NK + 3 - NZ*NK] \
                   + C[posc + 3] * C[posc + 3 - NZ*NK] * C[posc + NK + 3 - NZ*NK] \
                   + C[posc + 3] * C[posc + 3 - NZ*NK] * C[posc + NK + 3] );
      }

			txx[pos] += C[posc + 1] * dtdx * Dvxdx + C[posc + 2] * dtdz * Dvzdz;
		  tzz[pos] += C[posc + 2] * dtdx * Dvxdx + C[posc + 1] * dtdz * Dvzdz;
		  txz[pos] += Ctmp * ( Dvxdz * dtdz + Dvzdx * dtdx);


    //  __syncthreads();

			zcom =  ((ix >= nabs) && (ix < NX - nabs) && (iz >= NZ - nabs)) ? 1:0;
			xcom =  (ix <= nabs) ? 0: ( (ix < NX - nabs) ? (ix - nabs) : (NX - 2*nabs));
			pmlpos = pos - ( NZ - fnabs - nabs) * (zcom + xcom);
			// update pml
			if(ix < nabs) {
				ipml = nabs - 1 - ix;
				// txx & tzz
				Vtmp = (2.0 * pml_vxx[pmlpos] + pml_para[ipml*12+5] * Dvxdx) / (2.0 + pml_para[ipml*12+1]);
				txx[pos] = txx[pos] + C[posc + 1] * dtdx * ( (pml_para[ipml*12+3] - 1) * Dvxdx - pml_para[ipml*12+3] * Vtmp * dx );
				tzz[pos] = tzz[pos] + C[posc + 2] * dtdx * ( (pml_para[ipml*12+3] - 1) * Dvxdx - pml_para[ipml*12+3] * Vtmp * dx);
				pml_vxx[pmlpos] = 2.0 * Vtmp - pml_vxx[pmlpos];
				// txz
				Vtmp = (2.0 * pml_vzx[pmlpos] + pml_para[ipml*12+4] * Dvzdx) / (2.0 + pml_para[ipml*12+0]);
				txz[pos] = txz[pos] + Ctmp * dtdx * ( (pml_para[ipml*12+2] - 1) * Dvzdx - pml_para[ipml*12+2] * Vtmp * dx );
				pml_vzx[pmlpos] = 2.0 * Vtmp - pml_vzx[pmlpos];
			}
			if(ix >= NX - nabs) {
				ipml = ix - (NX - nabs);
				// txx & tzz
				Vtmp = (2.0 * pml_vxx[pmlpos] + pml_para[ipml*12+5] * Dvxdx) / (2.0 + pml_para[ipml*12+1]);
				txx[pos] = txx[pos] + C[posc + 1] * dtdx * ( (pml_para[ipml*12+3] - 1) * Dvxdx - pml_para[ipml*12+3] * Vtmp * dx );
				tzz[pos] = tzz[pos] + C[posc + 2] * dtdx * ( (pml_para[ipml*12+3] - 1) * Dvxdx - pml_para[ipml*12+3] * Vtmp * dx);
				pml_vxx[pmlpos] = 2.0 * Vtmp - pml_vxx[pmlpos];
			}
			if(ix > NX - nabs) {
				// txz
				ipml = ix - (NX - nabs) - 1;
				Vtmp = (2.0 * pml_vzx[pmlpos] + pml_para[ipml*12+4] * Dvzdx) / (2.0 + pml_para[ipml*12+0]);
				txz[pos] = txz[pos] + Ctmp * dtdx * ( (pml_para[ipml*12+2] - 1) * Dvzdx - pml_para[ipml*12+2] * Vtmp * dx );
				pml_vzx[pmlpos] = 2.0 * Vtmp - pml_vzx[pmlpos];
			}

			if(iz < fnabs) {
				ipml = fnabs - 1 - iz;
				// txx & tzz
				Vtmp = (2.0 * pml_vzz[pmlpos] + pml_para[ipml*12+10] * Dvzdz) / (2.0 + pml_para[ipml*12+6]);
				txx[pos] = txx[pos] + C[posc + 2] * dtdz * ( (pml_para[ipml*12+8] - 1) * Dvzdz - pml_para[ipml*12+8] * Vtmp * dz );
				tzz[pos] = tzz[pos] + C[posc + 1] * dtdz * ( (pml_para[ipml*12+8] - 1) * Dvzdz - pml_para[ipml*12+8] * Vtmp * dz );
				pml_vzz[pmlpos] = 2.0 * Vtmp - pml_vzz[pmlpos];
				// txz
				Vtmp = (2.0 * pml_vxz[pmlpos] + pml_para[ipml*12+11] * Dvxdz) / (2.0 + pml_para[ipml*12+7]);
				txz[pos] = txz[pos] + Ctmp * dtdz * ( (pml_para[ipml*12+9] - 1) * Dvxdz - pml_para[ipml*12+9] * Vtmp * dz );
				pml_vxz[pmlpos] = 2.0 * Vtmp - pml_vxz[pmlpos];
			}

			if(iz >= NZ - nabs) {
				// txz
				ipml = iz - (NZ - nabs);
				Vtmp = (2.0 * pml_vxz[pmlpos] + pml_para[ipml*12+11] * Dvxdz) / (2.0 + pml_para[ipml*12+7]);
				txz[pos] = txz[pos] + Ctmp * dtdz * ( (pml_para[ipml*12+9] - 1) * Dvxdz - pml_para[ipml*12+9] * Vtmp * dz );
				pml_vxz[pmlpos] = 2.0 * Vtmp - pml_vxz[pmlpos];
			}
			if(iz > NZ - nabs) {
			// txx & tzz
				ipml = iz - (NZ - nabs) - 1;
				Vtmp = (2.0 * pml_vzz[pmlpos] + pml_para[ipml*12+10] * Dvzdz) / (2.0 + pml_para[ipml*12+6]);
				txx[pos] = txx[pos] + C[posc + 2] * dtdz * ( (pml_para[ipml*12+8] - 1) * Dvzdz - pml_para[ipml*12+8] * Vtmp * dz );
				tzz[pos] = tzz[pos] + C[posc + 1] * dtdz * ( (pml_para[ipml*12+8] - 1) * Dvzdz - pml_para[ipml*12+8] * Vtmp * dz );
				pml_vzz[pmlpos] = 2.0 * Vtmp - pml_vzz[pmlpos];
			}

    //   __syncthreads();

  		}

	return;
}
